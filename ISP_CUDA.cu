#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include ""
#include <stdio.h>
#include <assert.h>

#define HANDLE_ERROR(A) assert(A == hipSuccess) 


__global__ void BayerProcess(int w, int h, unsigned short* in_img, float wb_r, float wb_b, int BlackLevel, unsigned short* out_img)
{
    int x = threadIdx.x + blockIdx.x * blockDim.x;//��ǰ��
    int y = threadIdx.y + blockIdx.y * blockDim.y;//��ǰ��


    int idx = x + y * w;

    out_img[idx] = in_img[idx] - BlackLevel;

    int xidx = x % 2;
    int yidx = y % 2;

    //R
    if ((xidx == 0) && (yidx == 0))
        out_img[idx] = out_img[idx] * wb_r;

    //B
    if ((xidx == 1) && (yidx == 1))
        out_img[idx] = out_img[idx] * wb_b;

}

__global__ void BayerGray(int w, int h, unsigned short* in_img, unsigned short* out_img)
{
    int x = threadIdx.x + blockIdx.x * blockDim.x;//��ǰ��
    int y = threadIdx.y + blockIdx.y * blockDim.y;//��ǰ��


    int idx = x + y * w;

    unsigned short pixel = in_img[idx];
    unsigned short* pout = out_img + (idx * 3);

    pout[0] = pixel;
    pout[1] = pixel;
    pout[2] = pixel;

    /*out_img[idx] = pixel;
    out_img[idx + 1] = pixel;
    out_img[idx + 2] = pixel;*/

}

__inline__ __device__ unsigned short saturate12bits(int value)
{
    if (value < 0)
        return 0;
    else if (value > 4095)
        return 4095;
    else
        return value;
}
__global__ void GPUMalvarDemosaicing(int w, int h, unsigned short* img_raw, unsigned short* img_data)
{
	int x = 2 * threadIdx.x + (blockIdx.x * blockDim.x << 1);
	int y = 2 * threadIdx.y + (blockIdx.y * blockDim.y << 1);

	int w3 = w * 3;

	int offsets[24] = { -w * 2,-w * 2 + 1,
					-w - 1,-w,-w + 1,-w + 2,
						 -2,-1,0,1,2,3,
				  w - 2,w - 1,w,w + 1,w + 2,w + 3,
				 w * 2 - 1,w * 2,w * 2 + 1,w * 2 + 2,
						w * 3,w * 3 + 1 };

	int centerIndex = x + y * w;
	int outIndex = (y * w + x) * 3;

	if ((x > 1) && (x < w - 1) && (y > 1) && (y < h - 1))
	{
		int indexTT1 = centerIndex + offsets[0];
		int indexTT2 = centerIndex + offsets[1];

		int indexT1 = centerIndex + offsets[2];
		int indexT2 = centerIndex + offsets[3];
		int indexT3 = centerIndex + offsets[4];
		int indexT4 = centerIndex + offsets[5];

		int indexLL = centerIndex + offsets[6];
		int indexL = centerIndex + offsets[7];
		int indexC = centerIndex + offsets[8];
		int indexR = centerIndex + offsets[9];
		int indexRR = centerIndex + offsets[10];
		int indexRRR = centerIndex + offsets[11];

		int indexBLL = centerIndex + offsets[12];
		int indexBL = centerIndex + offsets[13];
		int indexBC = centerIndex + offsets[14];
		int indexBR = centerIndex + offsets[15];
		int indexBRR = centerIndex + offsets[16];
		int indexBRRR = centerIndex + offsets[17];

		int indexBB1 = centerIndex + offsets[18];
		int indexBB2 = centerIndex + offsets[19];
		int indexBB3 = centerIndex + offsets[20];
		int indexBB4 = centerIndex + offsets[21];

		int indexBBB1 = centerIndex + offsets[22];
		int indexBBB2 = centerIndex + offsets[23];

		unsigned short R[6];
		unsigned short G[12];
		unsigned short B[6];

		R[0] = img_raw[indexTT1];
		R[1] = img_raw[indexLL];
		R[2] = img_raw[indexC];
		R[3] = img_raw[indexRR];
		R[4] = img_raw[indexBB2];
		R[5] = img_raw[indexBB4];

		G[0] = img_raw[indexTT2];
		G[1] = img_raw[indexT2];
		G[2] = img_raw[indexT4];
		G[3] = img_raw[indexL];
		G[4] = img_raw[indexR];
		G[5] = img_raw[indexRRR];
		G[6] = img_raw[indexBLL];
		G[7] = img_raw[indexBC];
		G[8] = img_raw[indexBRR];
		G[9] = img_raw[indexBB1];
		G[10] = img_raw[indexBB3];
		G[11] = img_raw[indexBBB1];

		B[0] = img_raw[indexT1];
		B[1] = img_raw[indexT3];
		B[2] = img_raw[indexBL];
		B[3] = img_raw[indexBR];
		B[4] = img_raw[indexBRRR];
		B[5] = img_raw[indexBBB2];

		unsigned short outR1, outG1, outB1;
		unsigned short outR2, outG2, outB2;
		unsigned short outR3, outG3, outB3;
		unsigned short outR4, outG4, outB4;

		int R1, G1, B1;
		int R2, G2, B2;
		int R3, G3, B3;
		int R4, G4, B4;

		R1 = R[2];
		G1 = ((R[2] << 2) - (R[0] + R[1] + R[3] + R[4]) + ((G[1] + G[3] + G[4] + G[7]) << 1)) >> 3;
		B1 = (((R[2] * 12) - ((R[0] + R[1] + R[3] + R[4]) * 3) + ((B[0] + B[1] + B[2] + B[3]) << 2)) >> 1) >> 3;

		R2 = (((R[2] + R[3]) << 2) - (G[1] + G[2] + G[3] + G[5] + G[7] + G[8]) + ((G[0] + G[10]) >> 1) + (G[4] * 5)) >> 3;
		G2 = G[4];
		B2 = (((B[1] + B[3]) << 2) - (G[0] + G[1] + G[2] + G[7] + G[8] + G[10]) + ((G[3] + G[5]) >> 1) + (G[4] * 5)) >> 3;

		R3 = (((R[2] + R[4]) << 2) - (G[1] + G[3] + G[4] + G[9] + G[10] + G[11]) + ((G[6] + G[8]) >> 1) + (G[7] * 5)) >> 3;
		G3 = G[7];
		B3 = (((B[2] + B[3]) << 2) - (G[3] + G[4] + G[6] + G[8] + G[9] + G[10]) + ((G[1] + G[11]) >> 1) + (G[7] * 5)) >> 3;

		R4 = ((((R[2] + R[3] + R[4] + R[5]) << 2) - ((B[1] + B[2] + B[4] + B[5]) * 3) + (B[3] * 12)) >> 1) >> 3;
		G4 = ((B[3] << 2) - (B[1] + B[2] + B[4] + B[5]) + ((G[4] + G[7] + G[8] + G[10]) << 1)) >> 3;
		B4 = B[3];

		outR1 = saturate12bits(R1);
		outR2 = saturate12bits(R2);
		outR3 = saturate12bits(R3);
		outR4 = saturate12bits(R4);
		outG1 = saturate12bits(G1);
		outG2 = saturate12bits(G2);
		outG3 = saturate12bits(G3);
		outG4 = saturate12bits(G4);
		outB1 = saturate12bits(B1);
		outB2 = saturate12bits(B2);
		outB3 = saturate12bits(B3);
		outB4 = saturate12bits(B4);

		img_data[outIndex] = outB1;
		img_data[outIndex + 1] = outG1;
		img_data[outIndex + 2] = outR1;

		img_data[outIndex + 3] = outB2;
		img_data[outIndex + 4] = outG2;
		img_data[outIndex + 5] = outR2;

		img_data[outIndex + w3] = outB3;
		img_data[outIndex + w3 + 1] = outG3;
		img_data[outIndex + w3 + 2] = outR3;

		img_data[outIndex + w3 + 3] = outB4;
		img_data[outIndex + w3 + 4] = outG4;
		img_data[outIndex + w3 + 5] = outR4;
	}
}

extern "C" hipError_t cudaISP(int w, int h, unsigned short* img_bayer, unsigned short* img_rgb, float* elapsedTime)
{
    unsigned short* dev_img_bayer = 0;
    unsigned short* dev_img_rgb = 0;
    hipError_t cudaStatus;

    dim3 blocks((w + 15) / 16, (h + 15) / 16);
    dim3 threads(16, 16);

	dim3 blocks2((w + 31) / 32, (h + 31) / 32);

    // Choose which GPU to run on, change this on a multi-GPU system.
    cudaStatus = hipSetDevice(0);
    HANDLE_ERROR(cudaStatus);

    // Allocate GPU buffers for three vectors (two input, one output)    .
    cudaStatus = hipMalloc((void**)&dev_img_bayer, w * h * sizeof(unsigned short));
    HANDLE_ERROR(cudaStatus);

    cudaStatus = hipMalloc((void**)&dev_img_rgb, w * h * sizeof(unsigned short) * 3);
    HANDLE_ERROR(cudaStatus);

    cudaStatus = hipMemcpy(dev_img_bayer, img_bayer, w * h * sizeof(unsigned short), hipMemcpyHostToDevice);
    HANDLE_ERROR(cudaStatus);

	//��ʼ��¼ʱ��
	hipEvent_t e_start, e_stop;
	hipEventCreate(&e_start);
	hipEventCreate(&e_stop);
	hipEventRecord(e_start, 0);

    // Launch a kernel on the GPU with one thread for each element.
    //BayerGray << <blocks, threads >> > (w, h, dev_img_bayer, dev_img_rgb);
    GPUMalvarDemosaicing << <blocks2, threads >> > (w, h, dev_img_bayer, dev_img_rgb);

    // Check for any errors launching the kernel
    cudaStatus = hipGetLastError();
    HANDLE_ERROR(cudaStatus);

    // hipDeviceSynchronize waits for the kernel to finish, and returns
    // any errors encountered during the launch.
    cudaStatus = hipDeviceSynchronize();
    HANDLE_ERROR(cudaStatus);

	//�����ʱ
	hipEventRecord(e_stop, 0);
	hipEventSynchronize(e_stop);
	hipEventElapsedTime(elapsedTime, e_start, e_stop);

	

    // Copy output vector from GPU buffer to host memory.
    cudaStatus = hipMemcpy(img_rgb, dev_img_rgb, w * h * sizeof(unsigned short) * 3, hipMemcpyDeviceToHost);
    HANDLE_ERROR(cudaStatus);

    cudaStatus = hipFree(dev_img_bayer);
    HANDLE_ERROR(cudaStatus);

    cudaStatus = hipFree(dev_img_rgb);
    HANDLE_ERROR(cudaStatus);

    return cudaStatus;
}